#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <hip/device_functions.h>
#include <time.h>
#include "curisk.cuh"

__global__ void generate_vector_sample_kernel();
__global__ void setup_gamma_generator(long seed);
__device__ __forceinline__ float generate_gamma_1_1(hiprandState *state);

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("\nError \"%s\" at %s:%d\n", hipGetErrorString(x), __FILE__, __LINE__);\
    exit(EXIT_FAILURE);}} while(0)

#define check_error() do { if(hipGetLastError()!=hipSuccess) { \
    printf("\nError \"%s\" at %s:%d\n", hipGetErrorString(hipGetLastError()), __FILE__, __LINE__);\
    exit(EXIT_FAILURE);}} while(0)

#define ESTIMATED_MAX_DIMENSION 32

__constant__ int c_dimension;
__constant__ int c_vector_scheme[ESTIMATED_MAX_DIMENSION];
__constant__ bound_t c_vector_bounds[ESTIMATED_MAX_DIMENSION];
__constant__ ordinal_t c_vector_ordinal[ESTIMATED_MAX_DIMENSION];
__constant__ int c_sample_size;

__device__ float *d_vector_sample;
__device__ int d_round_sample_size;
__device__ float *d_round_vector_sample;
__device__ hiprandState *d_curand_states;
__device__ int d_vectors_ready;

void generate_vector_sample(sampling_cofiguration_t& conf, sampling_result_t& result, int timeout_rounds)
{
    int start_time_point = clock();

    /* Скопируем некоторые поля из conf в константную память. */
    conf.log() << "Preparing constant variables." << std::endl;
    hipMemcpyToSymbol(HIP_SYMBOL(c_dimension), &conf.dimension, sizeof(int)); check_error();
    hipMemcpyToSymbol(HIP_SYMBOL(c_sample_size), &conf.sample_size, sizeof(int)); check_error();
    hipMemcpyToSymbol(HIP_SYMBOL(c_vector_scheme), conf.vector_scheme, conf.dimension*sizeof(int)); check_error();
    hipMemcpyToSymbol(HIP_SYMBOL(c_vector_bounds), conf.vector_bounds, conf.dimension*sizeof(bound_t)); check_error();
    hipMemcpyToSymbol(HIP_SYMBOL(c_vector_ordinal), conf.vector_ordinal, conf.dimension*sizeof(ordinal_t)); check_error();

    /* Выделим память для выборки. */
    conf.log() << "Allocate memory for vector sample." << std::endl;
    float *dh_vector_sample;
    hipMalloc(&dh_vector_sample, conf.sample_size*conf.dimension*sizeof(float)); check_error();
    hipMemcpyToSymbol(HIP_SYMBOL(d_vector_sample), &dh_vector_sample, sizeof(dh_vector_sample)); check_error();

    /* Выделим память для выборки раунда. */
    conf.log() << "Allocate memory for round vector sample." << std::endl;
    int blocks_per_round = conf.grid_dimension.x;
    int vectors_per_block = conf.block_dimension.x;
    int round_sample_size = blocks_per_round*vectors_per_block;
    hipMemcpyToSymbol(HIP_SYMBOL(d_round_sample_size), &round_sample_size, sizeof(int)); check_error();

    float *dh_round_vector_sample;
    hipMalloc(&dh_round_vector_sample, round_sample_size*conf.dimension*sizeof(float)); check_error();
    hipMemcpyToSymbol(HIP_SYMBOL(d_round_vector_sample), &dh_round_vector_sample, sizeof(dh_round_vector_sample)); check_error();

    /* Настроим генератор случайных чисел. */
    conf.log() << "Setup CUDA random numbers generator." << std::endl;
    hiprandState *dh_curand_states;
    hipMalloc(&dh_curand_states, round_sample_size*sizeof(hiprandState)); check_error();
    hipMemcpyToSymbol(HIP_SYMBOL(d_curand_states), &dh_curand_states, sizeof(dh_curand_states)); check_error();

    setup_gamma_generator<<<blocks_per_round, vectors_per_block>>>(clock()); check_error();
    hipDeviceSynchronize(); check_error();

    /* Число готовых элементов выборки. */
    int vectors_ready = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(d_vectors_ready), &vectors_ready, sizeof(vectors_ready)); check_error();

    conf.log() << "Start round cycle." << std::endl;
    int rounds = 0;
    while (vectors_ready < conf.sample_size)
    {
        generate_vector_sample_kernel<<<blocks_per_round, vectors_per_block>>>();
        CUDA_CALL(hipDeviceSynchronize());
        hipMemcpyFromSymbol(&vectors_ready, HIP_SYMBOL(d_vectors_ready), sizeof(vectors_ready)); check_error();
        rounds++;
        if (rounds > timeout_rounds)
        {
            conf.log() << "Round cycle is terminated (timeout)." << std::endl;
            result.generated_vectors_number = vectors_ready;
            result.error = SAMPLING_TIMEOUT;
            break;
        }
    }

    conf.log() << "Stop round cycle." << std::endl;
    conf.log() << "Vectors generated: " << vectors_ready << "/" << conf.sample_size << "." << std::endl;

    if (vectors_ready < conf.sample_size)
    {
        hipMemcpy(result.vector_sample, dh_vector_sample, vectors_ready*conf.dimension*sizeof(float), hipMemcpyDeviceToHost); check_error();
    }
    else
    {
        hipMemcpy(result.vector_sample, dh_vector_sample, conf.sample_size*conf.dimension*sizeof(float), hipMemcpyDeviceToHost); check_error();
    }
    
    hipFree(dh_vector_sample);
    hipFree(dh_round_vector_sample);
    hipFree(dh_curand_states);

    int end_time_point = clock();
    float elapsed_time = ((float) (end_time_point - start_time_point))/CLOCKS_PER_SEC;
    conf.log() << "Elapsed time: " << elapsed_time << " s." << std::endl;
    result.elapsed_time = elapsed_time;
}

__global__
void generate_vector_sample_kernel()
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int dimension = c_dimension;
    int i;
    float value;
    float than_value;
    float sum = 0;
    int last_vectors_ready;
    ordinal_t ordinal;
    bound_t bound;
    bool eliminate = false;

    for (i = 0; i < dimension; i++)
    {
        if (c_vector_scheme[i] == 0)
            value = 0;
        else
            value = generate_gamma_1_1(d_curand_states + idx);

        sum += value;
        d_round_vector_sample[dimension*idx + i] = value;
    }

    if (sum != 0)
    {
        for (i = 0; i < dimension; i++)
        {
            value = d_round_vector_sample[dimension*idx + i];
            value /= sum;
            d_round_vector_sample[dimension*idx + i] = value;
            bound = c_vector_bounds[i];
            eliminate = eliminate || value < bound.left || value > bound.right;
        }

        for (i = 0; i < dimension; i++)
        {
            value = d_round_vector_sample[dimension*idx + i];
            ordinal = c_vector_ordinal[i];
            than_value = d_round_vector_sample[dimension*idx + ordinal.than_index];
            eliminate = eliminate ||
                (ordinal.ordinal == ORDINAL_LESS && value >= than_value) ||
                (ordinal.ordinal == ORDINAL_MORE && value <= than_value);
        }
    }

    if (!eliminate)
    {
        last_vectors_ready = atomicAdd(&d_vectors_ready, 1);
        if (last_vectors_ready < c_sample_size)
        {
            for (i = 0; i < dimension; i++)
                d_vector_sample[dimension*last_vectors_ready + i] = d_round_vector_sample[dimension*idx + i];
        }
    }
}

__global__
void setup_gamma_generator(long seed)
{
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if (tid < d_round_sample_size)
        hiprand_init(seed, tid, 0, &d_curand_states[tid]);
}

/*
 * Функция генерирует случайную величину с распределением Gamma(1,1).
 */
__device__ __forceinline__
float generate_gamma_1_1(hiprandState *state)
{
    hiprandState localState = *state;
    float c, z, u, v, result;

    c = 1/sqrtf(9*2/3.);

    do {
        z = hiprand_normal(&localState);
        u = hiprand_uniform(&localState);
        v = powf(1 + c*z, 3);
    } while (z <= (-1/c) || logf(u) >= (0.5*z*z + 2/3. - (2/3.)*v + (2/3.)*logf(v)));

    result = (2/3.)*v;

    *state = localState;

    return result;
}