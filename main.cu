#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>

#include "curisk.cuh"
#include "nullbuf.cuh"

//std::ofstream logFile("log.txt", std::ofstream::out);
std::ostream& logger = std::cout;

#define THREADS_PER_BLOCK 128
#define BLOCKS_PER_GRID 64

void write_sample(std::ostream& output, sampling_cofiguration_t& conf, sampling_result_t& result, std::streamsize precision = 4)
{
    std::streamsize width = precision + 2;
    for (int i = 0; i < conf.sample_size; i++)
    {
        for (int j = 0; j < conf.dimension; j++)
        {
            float value = result.vector_sample[conf.dimension*i + j];
            output << std::fixed << std::setw(width) << std::setprecision(precision) 
                << std::setfill('0') << value << " ";
        }
        output << std::endl;
    }
}

void curisk_test_simple()
{
    logger << "Running curisk_test_simple." << std::endl;

    int vector_scheme[3] = {1, 1, 1};
    bound_t vector_bounds[3] = {BOUND_INIT, BOUND_INIT, BOUND_INIT};
    ordinal_t vector_ordinal[3] = {ORDINAL_INIT, ORDINAL_INIT, ORDINAL_INIT};

    sampling_cofiguration_t conf(3, 1000, dim3(BLOCKS_PER_GRID, 1, 1), dim3(THREADS_PER_BLOCK, 1, 1));
    memcpy(conf.vector_scheme, vector_scheme, sizeof(vector_scheme));
    memcpy(conf.vector_bounds, vector_bounds, sizeof(vector_bounds));
    memcpy(conf.vector_ordinal, vector_ordinal, sizeof(vector_ordinal));
    sampling_result_t result(&conf);
    generate_vector_sample(conf, result, 5000);

    std::ofstream output("curisk_test_simple.txt", std::ofstream::out);
    write_sample(output, conf, result);
    output.close();

    logger << std::endl;
}

void curisk_test_bounds()
{
    logger << "Running curisk_test_bounds." << std::endl;

    int vector_scheme[3] = {1, 1, 1};
    bound_t vector_bounds[3] = {{0.2, 0.7}, {0.1, 0.8}, {0.5, 0.9}};
    ordinal_t vector_ordinal[3] = {ORDINAL_INIT, ORDINAL_INIT, ORDINAL_INIT};

    sampling_cofiguration_t conf(3, 1000, dim3(BLOCKS_PER_GRID, 1, 1), dim3(THREADS_PER_BLOCK, 1, 1));
    memcpy(conf.vector_scheme, vector_scheme, sizeof(vector_scheme));
    memcpy(conf.vector_bounds, vector_bounds, sizeof(vector_bounds));
    memcpy(conf.vector_ordinal, vector_ordinal, sizeof(vector_ordinal));
    sampling_result_t result(&conf);
    generate_vector_sample(conf, result, 5000);

    std::ofstream output("curisk_test_bounds.txt", std::ofstream::out);
    write_sample(output, conf, result);
    output.close();

    logger << std::endl;
}

void curisk_test_ordinal()
{
    logger << "Running curisk_test_ordinal." << std::endl;

    int vector_scheme[3] = {1, 1, 1};
    bound_t vector_bounds[3] = {BOUND_INIT, BOUND_INIT, BOUND_INIT};
    ordinal_t vector_ordinal[3] = {
        {2, ORDINAL_MORE},
        {2, ORDINAL_LESS},
        ORDINAL_INIT
    };

    sampling_cofiguration_t conf(3, 1000, dim3(BLOCKS_PER_GRID, 1, 1), dim3(THREADS_PER_BLOCK, 1, 1));
    memcpy(conf.vector_scheme, vector_scheme, sizeof(vector_scheme));
    memcpy(conf.vector_bounds, vector_bounds, sizeof(vector_bounds));
    memcpy(conf.vector_ordinal, vector_ordinal, sizeof(vector_ordinal));
    sampling_result_t result(&conf);
    generate_vector_sample(conf, result, 5000);

    std::ofstream output("curisk_test_ordinal.txt", std::ofstream::out);
    write_sample(output, conf, result);
    output.close();

    logger << std::endl;
}

void curisk_test_ordinal_and_bounds()
{
    logger << "Running curisk_test_ordinal_and_bounds." << std::endl;

    int vector_scheme[3] = {1, 1, 1};
    bound_t vector_bounds[3] = {{0.2, 0.3}, {0.1, 0.8}, {0.5, 0.9}};
    ordinal_t vector_ordinal[3] = {
        {1, ORDINAL_MORE},
        {2, ORDINAL_LESS},
        ORDINAL_INIT
    };

    sampling_cofiguration_t conf(3, 1000, dim3(BLOCKS_PER_GRID, 1, 1), dim3(THREADS_PER_BLOCK, 1, 1));
    memcpy(conf.vector_scheme, vector_scheme, sizeof(vector_scheme));
    memcpy(conf.vector_bounds, vector_bounds, sizeof(vector_bounds));
    memcpy(conf.vector_ordinal, vector_ordinal, sizeof(vector_ordinal));
    sampling_result_t result(&conf);
    generate_vector_sample(conf, result, 5000);

    std::ofstream output("curisk_test_ordinal_and_bounds.txt", std::ofstream::out);
    write_sample(output, conf, result);
    output.close();

    logger << std::endl;
}

void curisk_test_bounds_increase()
{
    // Будем увеличивать левую границу второго элемента.
    logger << "Running curisk_test_bounds_increase." << std::endl;

    int vector_scheme[3] = {1, 1, 1};
    bound_t vector_bounds[3] = {{0.2, 0.3}, {0.1, 0.8}, {0.5, 0.9}};
    ordinal_t vector_ordinal[3] = {ORDINAL_INIT, ORDINAL_INIT, ORDINAL_INIT};

    sampling_cofiguration_t conf(3, 1000, dim3(BLOCKS_PER_GRID, 1, 1), dim3(THREADS_PER_BLOCK, 1, 1));
    memcpy(conf.vector_scheme, vector_scheme, sizeof(vector_scheme));
    memcpy(conf.vector_bounds, vector_bounds, sizeof(vector_bounds));
    memcpy(conf.vector_ordinal, vector_ordinal, sizeof(vector_ordinal));
    sampling_result_t result(&conf);

    float left_bound = 0.1;
    float step = 0.001;
    int times = 10;
    std::ofstream output("curisk_test_bounds_increase.txt", std::ofstream::out);

    do
    {
        float total_time = 0;
        float average_time = 0;
        for (int i = 0; i < times; i++)
        {
            conf.vector_bounds[1].left = left_bound;
            generate_vector_sample(conf, result, 5000);
            if (result.error == SAMPLING_TIMEOUT)
                break;
            total_time += result.elapsed_time;
        }
        average_time = total_time/times;
        output << left_bound << " " << average_time << std::endl;
        left_bound += step;
    }
    while (result.error != SAMPLING_TIMEOUT);

    output.close();

    logger << std::endl;
}

void curisk_test_time()
{
    logger << "Running curisk_test_time." << std::endl;

    int vector_scheme[10] = {1, 1, 1, 1, 1, 1, 1, 1, 1, 1};
    bound_t vector_bounds[10] = {
        BOUND_INIT, BOUND_INIT, BOUND_INIT, BOUND_INIT, BOUND_INIT,
        BOUND_INIT, BOUND_INIT, BOUND_INIT, BOUND_INIT, BOUND_INIT,
    };
    ordinal_t vector_ordinal[10] = {
        ORDINAL_INIT, ORDINAL_INIT, ORDINAL_INIT, ORDINAL_INIT, ORDINAL_INIT,
        ORDINAL_INIT, ORDINAL_INIT, ORDINAL_INIT, ORDINAL_INIT, ORDINAL_INIT
    };

    int n_vectors = 1000000;

    sampling_cofiguration_t conf(10, n_vectors, dim3(BLOCKS_PER_GRID, 1, 1), dim3(THREADS_PER_BLOCK, 1, 1), cnull);
    memcpy(conf.vector_scheme, vector_scheme, sizeof(vector_scheme));
    memcpy(conf.vector_bounds, vector_bounds, sizeof(vector_bounds));
    memcpy(conf.vector_ordinal, vector_ordinal, sizeof(vector_ordinal));
    sampling_result_t result(&conf);

    int times = 30;
    float total = 0;
    float average = 0;
    for (int i = 0; i < times; i++)
    {
        generate_vector_sample(conf, result, 5000);
        total += result.elapsed_time;
    }
    average = total/times;

    logger << "Dimension: " << 10 << "." << std::endl;
    logger << "Generated vectors: " << n_vectors << "." << std::endl; 
    logger << "Average time: " << average << " s." << std::endl; 
    logger << std::endl;
}

int main(int argc, char *argv[])
{
    /*
     * При первом запуске ядер CUDA производится их компиляция в Runtime.
     * Это занимает сравнителньо много времени, поэтому запустим один тест
     * вхолостую.
     */
    curisk_test_simple();

    curisk_test_simple();
    curisk_test_bounds();
    curisk_test_ordinal();
    curisk_test_ordinal_and_bounds();
    //curisk_test_bounds_increase();
    curisk_test_time();

    hipDeviceReset();

    return EXIT_SUCCESS;
}